
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>

#define DELTA 0.02
#define ALPHA 1.0
#define BETA 5.0
#define GAMMA 8.0
#define OMEGA 0.5

struct State
{
    double x;
    double y;
};

// Duffing egyenlet CUDA device függvény
__device__ State duffing_rhs(State state, double t)
{
    State ds;
    ds.x = state.y;
    ds.y = -DELTA * state.y - ALPHA * state.x - BETA * state.x * state.x * state.x + GAMMA * cos(OMEGA * t);
    return ds;
}

// RK4 CUDA device függvény
__device__ State rk4_step(State state, double t, double dt)
{
    State k1 = duffing_rhs(state, t);
    State k2 = duffing_rhs({state.x + dt * k1.x / 2, state.y + dt * k1.y / 2}, t + dt / 2);
    State k3 = duffing_rhs({state.x + dt * k2.x / 2, state.y + dt * k2.y / 2}, t + dt / 2);
    State k4 = duffing_rhs({state.x + dt * k3.x, state.y + dt * k3.y}, t + dt);

    State next_state;
    next_state.x = state.x + dt * (k1.x + 2 * k2.x + 2 * k3.x + k4.x) / 6;
    next_state.y = state.y + dt * (k1.y + 2 * k2.y + 2 * k3.y + k4.y) / 6;
    return next_state;
}

// CUDA kernel: minden szál egy initial_x-hez tartozó szimulációt futtat
__global__ void run_simulation_kernel(
    double *initial_xs, double *phase_x, double *phase_y,
    int num_initials, int num_steps, double dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_initials)
        return;

    double x0 = initial_xs[idx];
    State state = {x0, 0.0};
    double t = 0.0;

    for (int step = 0; step < num_steps; ++step)
    {
        state = rk4_step(state, t, dt);
        phase_x[idx * num_steps + step] = state.x;
        phase_y[idx * num_steps + step] = state.y;
        t += dt;
    }
}

int main()
{
    int num_initials = 1000;
    int num_steps = 1000; // például 100.0 idő, 0.01 lépésköz
    double dt = 0.01;

    // Host oldali adatok
    double *h_initial_xs = new double[num_initials];
    double *h_phase_x = new double[num_initials * num_steps];
    double *h_phase_y = new double[num_initials * num_steps];

    for (int i = 0; i < num_initials; ++i)
        h_initial_xs[i] = 0.1 + 0.05 * i;

    // Device oldali adatok
    double *d_initial_xs, *d_phase_x, *d_phase_y;
    hipMalloc(&d_initial_xs, num_initials * sizeof(double));
    hipMalloc(&d_phase_x, num_initials * num_steps * sizeof(double));
    hipMalloc(&d_phase_y, num_initials * num_steps * sizeof(double));

    hipMemcpy(d_initial_xs, h_initial_xs, num_initials * sizeof(double), hipMemcpyHostToDevice);

    // Időmérés kezdete
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Kernel indítása
    int blockSize = 128;
    int gridSize = (num_initials + blockSize - 1) / blockSize;
    run_simulation_kernel<<<gridSize, blockSize>>>(d_initial_xs, d_phase_x, d_phase_y, num_initials, num_steps, dt);
    hipDeviceSynchronize();

    // Időmérés vége
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel futási ideje: " << milliseconds << " ms" << std::endl;

    // Szálak és szimulációk összefoglalása
    std::cout << "Összefoglaló:" << std::endl;
    std::cout << "  Szálak száma (blockSize): " << blockSize << std::endl;
    std::cout << "  Grid méret (gridSize): " << gridSize << std::endl;
    std::cout << "  Összes indított CUDA szál: " << (blockSize * gridSize) << std::endl;
    std::cout << "  Szimulációk száma: " << num_initials << std::endl;
    std::cout << "  Egy szimuláció lépései: " << num_steps << std::endl;
    std::cout << "  Teljes futási idő: " << milliseconds << " ms" << std::endl;

    // Eredmények visszamásolása
    hipMemcpy(h_phase_x, d_phase_x, num_initials * num_steps * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_phase_y, d_phase_y, num_initials * num_steps * sizeof(double), hipMemcpyDeviceToHost);

    // Kiírás fájlba (kezdeti feltételek szerint, üres sorokkal elválasztva)
    std::ofstream fout("duffing_cuda_phase.csv");
    for (int i = 0; i < num_initials; ++i)
    {
        fout << "# initial_x = " << h_initial_xs[i] << "\n";
        fout << "x,y\n";
        for (int step = 0; step < num_steps; ++step)
        {
            fout << h_phase_x[i * num_steps + step] << "," << h_phase_y[i * num_steps + step] << "\n";
        }
        fout << "\n\n";
    }
    fout.close();

    // Felszabadítás
    hipFree(d_initial_xs);
    hipFree(d_phase_x);
    hipFree(d_phase_y);
    delete[] h_initial_xs;
    delete[] h_phase_x;
    delete[] h_phase_y;

    return 0;
}